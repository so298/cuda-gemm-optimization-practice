#include "hip/hip_runtime.h"
#include <iostream>

#include "common.hh"
#include "common_gpu.cuh"

__global__ void kernel_gpu_naive(size_t N, size_t K, size_t M, GemmBench::T* A,
                                 GemmBench::T* B, GemmBench::T* C) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  size_t x = idx / M;  // row idx of C
  size_t y = idx % M;  // col idx of C

  if (x < M && y < N) {
    GemmBench::T sum = 0;
    for (size_t k = 0; k < K; k++) {
      sum += A[x * K + k] * B[k * M + y];
    }
    C[x * M + y] = sum;
  }
}

class GemmBenchNaive : public GemmBench {
 public:
  void run(int num_iter, size_t N, size_t K, size_t M, T* A, T* B, T* C) {
    size_t block_size = 256;
    size_t grid_size = (N * M + block_size - 1) / block_size;

    for (int i = 0; i < num_iter; i++) {
      kernel_gpu_naive<<<grid_size, block_size>>>(N, K, M, A, B, C);
      CUDA_CHECK(hipDeviceSynchronize());
    }
  }
};

int main(int argc, char** argv) {
  Options opts(argc, argv);
  GemmBenchNaive bench;

  if (opts.test) {
    test_benchmark(bench, opts.N, opts.K, opts.M);
    return 0;
  }

  run_benchmark(bench, opts.num_iter, opts.N, opts.K, opts.M);
}